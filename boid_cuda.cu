#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <stdint.h>
#include <error.h>
#include <limits.h>
#include <pthread.h>
#include <time.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "boid_rules.h"

int SIZE = 10000;

#define PERCEPTION_RANGE 1000
#define AVOIDANCE_RANGE 100

#define COHERENCE_RATE 0.01
#define AVOIDANCE_RATE 1.0
#define ALIGNMENT_RATE 0.125

#define SCAN_BLOCK_DIM 256

#define DEBUG false

float *newLocation;
float *newVelocity;

float *oldLocation;
float *oldVelocity;

struct cudaDeviceInfo {
    float* oldLocation;
    float* oldVelocity;
    float* newLocation;
    float* newVelocity;
};

__device__ cudaDeviceInfo cudaData;

__device__ float deviceDist2(int x1, int y1, int x2, int y2){
    return ((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2));
}

__global__ void kernelRule1(cudaDeviceInfo cudaData, int SIZE) {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int localId = threadIdx.x;
    int workPerCycle = SCAN_BLOCK_DIM;
    int workCycles = (SIZE + SCAN_BLOCK_DIM - 1) / SCAN_BLOCK_DIM;
    float thisX, thisY;
    if (globalId < SIZE){
        thisX =  cudaData.oldLocation[globalId * 2];
        thisY = cudaData.oldLocation[globalId * 2 + 1];
    }
    float averageX = 0;
    float averageY = 0;
    int count = 0;

    __shared__ float2 sharedLocation[SCAN_BLOCK_DIM];
    for (int i = 0; i < workCycles; i++){
        int loadId = localId + i * workPerCycle;
        if (loadId < SIZE){
            sharedLocation[localId] = *(float2 *)&cudaData.oldLocation[loadId * 2];
        }
        __syncthreads();
        if (globalId < SIZE){
            for (int j = 0; j < workPerCycle; j++){
                if (globalId != j + workPerCycle * i && j + workPerCycle * i < SIZE){
                    float thatX = ((float *)sharedLocation)[j * 2];
                    float thatY = ((float *)sharedLocation)[j * 2 + 1];
                    if (deviceDist2(thisX, thisY, thatX, thatY) < PERCEPTION_RANGE * PERCEPTION_RANGE){
                        averageX += thatX;
                        averageY += thatY;
                        count += 1;
                    }
                }
            }
        }
    }
    averageX /= count;
    averageY /= count;
    if (globalId < SIZE){
        cudaData.newVelocity[globalId * 2] += (averageX - thisX) * COHERENCE_RATE;
        cudaData.newVelocity[globalId * 2 + 1] += (averageY - thisY) * COHERENCE_RATE;
    }
}

__global__ void kernelRule2(cudaDeviceInfo cudaData, int SIZE) {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int localId = threadIdx.x;
    int workPerCycle = SCAN_BLOCK_DIM;
    int workCycles = (SIZE + SCAN_BLOCK_DIM - 1) / SCAN_BLOCK_DIM;
    float thisX, thisY;
    if (globalId < SIZE){
        thisX =  cudaData.oldLocation[globalId * 2];
        thisY = cudaData.oldLocation[globalId * 2 + 1];
    }
    float avoidX = 0;
    float avoidY = 0;

    __shared__ float2 sharedLocation[SCAN_BLOCK_DIM];
    for (int i = 0; i < workCycles; i++){
        int loadId = localId + i * workPerCycle;
        if (loadId < SIZE){
            sharedLocation[localId] = *(float2 *)&cudaData.oldLocation[loadId * 2];
        }
        __syncthreads();
        if (globalId < SIZE){
            for (int j = 0; j < workPerCycle; j++){
                if (globalId != j + workPerCycle * i && j + workPerCycle * i < SIZE){
                    float thatX = ((float *)sharedLocation)[j * 2];
                    float thatY = ((float *)sharedLocation)[j * 2 + 1];
                    if (deviceDist2(thisX, thisY, thatX, thatY) < AVOIDANCE_RANGE * AVOIDANCE_RANGE){
                        avoidX -= (thatX - thisX);
                        avoidY -= (thatY - thisY);
                    }
                }
            }
        }
    }
    if (globalId < SIZE){
        cudaData.newVelocity[globalId * 2] += avoidX * AVOIDANCE_RATE;
        cudaData.newVelocity[globalId * 2 + 1] += avoidY * AVOIDANCE_RATE;
    }
}

__global__ void kernelRule3(cudaDeviceInfo cudaData, int SIZE) {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int localId = threadIdx.x;
    int workPerCycle = SCAN_BLOCK_DIM;
    int workCycles = (SIZE + SCAN_BLOCK_DIM - 1) / SCAN_BLOCK_DIM;
    float thisX, thisY;
    if (globalId < SIZE){
        thisX =  cudaData.oldLocation[globalId * 2];
        thisY = cudaData.oldLocation[globalId * 2 + 1];
    }
    float averageVelX = 0;
    float averageVelY = 0;
    int count = 0;

    __shared__ float2 sharedLocation[SCAN_BLOCK_DIM];
    __shared__ float2 sharedVelocity[SCAN_BLOCK_DIM];
    for (int i = 0; i < workCycles; i++){
        int loadId = localId + i * workPerCycle;
        if (loadId < SIZE){
            sharedLocation[localId] = *(float2 *)&cudaData.oldLocation[loadId * 2];
            sharedVelocity[localId] = *(float2 *)&cudaData.oldVelocity[loadId * 2];
        }
        __syncthreads();
        if (globalId < SIZE){
            for (int j = 0; j < workPerCycle; j++){
                if (globalId != j + workPerCycle * i && j + workPerCycle * i < SIZE){
                    float thatX = ((float *)sharedLocation)[j * 2];
                    float thatY = ((float *)sharedLocation)[j * 2 + 1];
                    if (deviceDist2(thisX, thisY, thatX, thatY) < AVOIDANCE_RANGE * AVOIDANCE_RANGE){
                        averageVelX += ((float *)sharedVelocity)[j * 2];
                        averageVelY += ((float *)sharedVelocity)[j * 2 + 1];
                        count ++;
                    }
                }
            }
        }
    }
    averageVelX /= count;
    averageVelY /= count;
    if (globalId < SIZE){
        cudaData.newVelocity[globalId * 2] += (averageVelX - cudaData.oldVelocity[globalId * 2]) * ALIGNMENT_RATE;
        cudaData.newVelocity[globalId * 2 + 1] += (averageVelY - cudaData.oldVelocity[globalId * 2 + 1]) * ALIGNMENT_RATE;
    }
}

__global__ void kernelUpdateLoc(cudaDeviceInfo cudaData, int SIZE) {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalId < SIZE){
        cudaData.newLocation[globalId * 2] += cudaData.newVelocity[globalId * 2];
        cudaData.newLocation[globalId * 2 + 1] += cudaData.newVelocity[globalId * 2 + 1];
    }
}

__global__ void kernelNew2Old(cudaDeviceInfo cudaData, int SIZE) {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalId < SIZE){
        cudaData.oldLocation[globalId * 2] = cudaData.newLocation[globalId * 2];
        cudaData.oldLocation[globalId * 2 + 1] = cudaData.newLocation[globalId * 2 + 1];
        cudaData.oldVelocity[globalId * 2] = cudaData.newVelocity[globalId * 2];
        cudaData.oldVelocity[globalId * 2 + 1] = cudaData.newVelocity[globalId * 2 + 1];
    }
}

void update() {
    int numBlocks = (SIZE + SCAN_BLOCK_DIM - 1) / SCAN_BLOCK_DIM;
    int blockSize = SCAN_BLOCK_DIM;
    kernelRule1<<<numBlocks, blockSize>>>(cudaData, SIZE);
    hipDeviceSynchronize();
    if (DEBUG) printf("Error at rule 1: %s\n", hipGetErrorString(hipGetLastError()));
    kernelRule2<<<numBlocks, blockSize>>>(cudaData, SIZE);
    hipDeviceSynchronize();
    if (DEBUG) printf("Error at rule 2: %s\n", hipGetErrorString(hipGetLastError()));
    kernelRule3<<<numBlocks, blockSize>>>(cudaData, SIZE);
    hipDeviceSynchronize();
    if (DEBUG) printf("Error at rule 3: %s\n", hipGetErrorString(hipGetLastError()));
    kernelUpdateLoc<<<numBlocks, blockSize>>>(cudaData, SIZE);
    hipDeviceSynchronize();
    if (DEBUG) printf("Error at update loc: %s\n", hipGetErrorString(hipGetLastError()));
    kernelNew2Old<<<numBlocks, blockSize>>>(cudaData, SIZE);
    hipDeviceSynchronize();
    if (DEBUG) printf("Error at copy loc/vel: %s\n", hipGetErrorString(hipGetLastError()));
}
void updateLinear() {
    for (int i = 0; i < SIZE; i++){
        rule1(i, oldLocation, newLocation, oldVelocity, newVelocity, SIZE);
        rule2(i, oldLocation, newLocation, oldVelocity, newVelocity, SIZE);
        rule3(i, oldLocation, newLocation, oldVelocity, newVelocity, SIZE);
        newLocation[i * 2] = oldLocation[i * 2] + newVelocity[i * 2];
        newLocation[i * 2 + 1] = oldLocation[i * 2 + 1] + newVelocity[i * 2 + 1];
    }
    memcpy(oldLocation, newLocation, sizeof(float) * 2 * SIZE);
    memcpy(oldVelocity, newVelocity, sizeof(float) * 2 * SIZE);
}

void setupCuda() {
    // declare device memory and copy data to device memory
    float *cudaDeviceOldLocation;
    float *cudaDeviceOldVelocity;
    float *cudaDeviceNewLocation;
    float *cudaDeviceNewVelocity;
    hipMalloc(&cudaDeviceOldLocation, sizeof(float) * 2 * SIZE);
    hipMalloc(&cudaDeviceNewLocation, sizeof(float) * 2 * SIZE);
    hipMalloc(&cudaDeviceOldVelocity, sizeof(float) * 2 * SIZE);
    hipMalloc(&cudaDeviceNewVelocity, sizeof(float) * 2 * SIZE);
    hipMemcpy(cudaDeviceOldLocation, oldLocation, sizeof(float) * 2 * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceNewLocation, newLocation, sizeof(float) * 2 * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceOldVelocity, oldVelocity, sizeof(float) * 2 * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceNewVelocity, newVelocity, sizeof(float) * 2 * SIZE, hipMemcpyHostToDevice);
    cudaData.oldLocation = cudaDeviceOldLocation;
    cudaData.newLocation = cudaDeviceNewLocation;
    cudaData.oldVelocity = cudaDeviceOldVelocity;
    cudaData.newVelocity = cudaDeviceNewVelocity;
    if (DEBUG) printf("Error at setup: %s\n", hipGetErrorString(hipGetLastError()));
}

void cleanup() {
    free(oldLocation);
    free(oldVelocity);
    free(newLocation);
    free(newVelocity);
    hipFree(cudaData.oldLocation);
    hipFree(cudaData.newLocation);
    hipFree(cudaData.oldVelocity);
    hipFree(cudaData.newVelocity);
}

int main(int argc, char **argv) {
    // NCORES = atoi(argv[2]);
    int iterations = 1;
    if (argc >= 3)
        iterations = atoi(argv[2]);
    if (argc >= 5)
        SIZE = atoi(argv[4]);
    struct timespec before, after;
    oldLocation = (float *)malloc(sizeof(float) * 2 * SIZE);
    oldVelocity = (float *)malloc(sizeof(float) * 2 * SIZE);
    newLocation = (float *)malloc(sizeof(float) * 2 * SIZE);
    newVelocity = (float *)malloc(sizeof(float) * 2 * SIZE);
    setup(oldLocation, newLocation, oldVelocity, newVelocity, SIZE);
    setupCuda();
    double average_ms = 0;
    for(int i = 0; i < iterations; i++){
        clock_gettime(CLOCK_REALTIME, &before);
            update();
        clock_gettime(CLOCK_REALTIME, &after);
        double delta_ms = (double)(after.tv_sec - before.tv_sec) * 1000.0 + (after.tv_nsec - before.tv_nsec) / 1000000.0;
        average_ms += delta_ms;
    }
    average_ms /= iterations;
    double cuda_time = average_ms;
    printf("Total time with cuda: %.3lf ms\n", average_ms);
    cleanup();

    oldLocation = (float *)malloc(sizeof(float) * 2 * SIZE);
    oldVelocity = (float *)malloc(sizeof(float) * 2 * SIZE);
    newLocation = (float *)malloc(sizeof(float) * 2 * SIZE);
    newVelocity = (float *)malloc(sizeof(float) * 2 * SIZE);
    setup(oldLocation, newLocation, oldVelocity, newVelocity, SIZE);
    setupCuda();
    average_ms = 0;
    for(int i = 0; i < iterations; i++){
        clock_gettime(CLOCK_REALTIME, &before);
            updateLinear();
        clock_gettime(CLOCK_REALTIME, &after);
        double delta_ms = (double)(after.tv_sec - before.tv_sec) * 1000.0 + (after.tv_nsec - before.tv_nsec) / 1000000.0;
        average_ms += delta_ms;
    }
    average_ms /= iterations;
    double normal_time = average_ms;
    printf("Total time linear: %.3lf ms\n", average_ms);
    printf("Speedup: %0.3f\n", normal_time / cuda_time);
    cleanup();
}